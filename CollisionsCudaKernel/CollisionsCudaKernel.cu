#include "hip/hip_runtime.h"
#include "CollisionsCudaKernel.h"
#include <windows.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "hiprand/hiprand_kernel.h"

__device__ double GenerateRandomNumber(int seed) {
	int result;
	double y;
	hiprandState_t state;
	int tid = blockIdx.x;

	hiprand_init(seed * tid, 0, tid, &state);

	result = hiprand(&state) % 100;

	y = (double) 1 / ((result + 1));

	return y;
}
__global__ void CollisionsKernel(int *dev_isParticlePassedThroughTheWall, double *dev_debug, int maxParticles, double rBasic, double sigma, double sigmaS, double wallThickness) {
	double pi = 3.14;

	int tid = blockIdx.x;

	if (tid < maxParticles) {
		// Cycle that emulates one particle lifecycle
		for (int i = 0; ; i++) {
			// Generate random number
			double y = GenerateRandomNumber(i);
			dev_debug[tid] = y;

			// Calculate random CosFi
			double cosFi = cos(2 * pi * y);
			if (cosFi < 0) {
				cosFi = cosFi * (-1);
			}

			// Calculate distance to the wall (r) if this is the first run
			double r;
			if (i == 0) {
				r = rBasic / cosFi;
			}
			else {
				r = 0;
			}

			// Calculate Epsilon
			double epsilonX;
			epsilonX = (-1 * (1 / sigma) * log(y) + r) * cosFi;
			if (i != 0) {
				epsilonX += rBasic;
			}

			// Determine is collision is inside the wall
			if (epsilonX >= rBasic && epsilonX <= rBasic + wallThickness) {
				// That means that the collision happened in the wall
				// Now we need to "calculate" fate of the particle
				if (y < sigmaS / sigma) {
					// In this case particle was absorbed
					// We will not count this particle
					dev_isParticlePassedThroughTheWall[tid] = 0;
					break;
				}
				else {
					// In this case particle wasn't absorbed, so we can run another itteration of particle lifecycle
				}
			}
			if (epsilonX < rBasic) {
				// In this case particle left the wall on the source side
				// We will not count this particle
				dev_isParticlePassedThroughTheWall[tid] = 0;
				break;
			}
			if (epsilonX > rBasic + wallThickness) {
				// In this case particle has passed through the wall
				// We will count this particle
				dev_isParticlePassedThroughTheWall[tid] = 1;
				break;
			}
		}
	}
}

int CallCollisionsCudaKernel(int maxParticles, double rBasic, double sigma, double sigmaS, double wallThickness) {
	int *arrayOfParticlesStatuses = new int [maxParticles];
	int *dev_arrayOfParticlesStatuses;

	double *debug = new double [maxParticles];
	double *dev_debug;

	hipMalloc((void**)&dev_arrayOfParticlesStatuses, maxParticles * sizeof(int));
	hipMalloc((void**)&dev_debug, maxParticles * sizeof(double));

	CollisionsKernel<<<maxParticles, 1>>> (dev_arrayOfParticlesStatuses, dev_debug, maxParticles, rBasic, sigma, sigmaS, wallThickness);
	
	hipMemcpy(arrayOfParticlesStatuses, dev_arrayOfParticlesStatuses, maxParticles * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_arrayOfParticlesStatuses);

	hipMemcpy(debug, dev_debug, maxParticles * sizeof(double), hipMemcpyDeviceToHost);
	hipFree(dev_debug);

	int countOfParticlesPassedThroughTheWall = 0;
	FILE *f;
	fopen_s(&f, "Result.dat", "a");
	fprintf_s(f, "Wall Thickness: %f, ", wallThickness);
	for (int i = 0; i < maxParticles; i++) {
		if (arrayOfParticlesStatuses[i] == 1) {
			countOfParticlesPassedThroughTheWall++;
		}
		/*fprintf_s(f, "Thread ID: %d, Particle passed through the wall: %d\n", i, arrayOfParticlesStatuses[i]);*/
	}

	int passRate = (double) countOfParticlesPassedThroughTheWall / maxParticles * 100;
	fprintf_s(f, "PassRate: %d\n", passRate);
	fclose(f);

	return countOfParticlesPassedThroughTheWall;
}